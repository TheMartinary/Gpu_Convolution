#include "hip/hip_runtime.h"
#include "convolution.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "logging.h"

typedef struct {
    int d;
    int h;
    int w;
} blob_dims;

//add padding to blob
BLOB* pad(BLOB* in, int pad){

    //create output blob
    BLOB* out = blob_calloc(in->d, in->h+2*pad, in->w+pad*2);

    //copy non-padded input into output blob
    for(int z=0;z<in->d;z++)
       for(int y=0;y<in->h;y++)
          for(int x=0;x<in->w;x++)
              blob_data(out,z,y+pad,x+pad)= blob_data(in,z,y,x);

    //return pointer to padded blob
    return out;
}

int get_next_pow2(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;

}

BLOB* load_weights(BLOB* b, conv_param_t* conv_param){

    //open weights file for reading
    info(conv_param->weights);
    info("\n");
    
    FILE* fp = fopen(conv_param->weights, "rb");
    if(fp==NULL)
        error("could not open file %s for reading\n",conv_param->weights);

    //for fully connected layers the kernel size is equal to the input size
    int Ky=(conv_param->fc)?b->h:conv_param->Ky;
    int Kx=(conv_param->fc)?b->w:conv_param->Kx;

    //allocate 3D blob, and emulate 4D in KxKy later
    BLOB* w = blob_alloc(conv_param->num_out, b->d/conv_param->group, Ky*Kx);

    //fill 4D weight structure
    for(int group_id=0;group_id<conv_param->group;group_id++)
        for(int out_depth=group_id*(conv_param->num_out/conv_param->group);out_depth<(group_id+1)*(conv_param->num_out/conv_param->group);out_depth++)
            for(int i=group_id*(b->d/conv_param->group);i<(group_id+1)*(b->d/conv_param->group);i++)
                //note: each output map has only  b->d/conv_param->group input maps. Hence the absolute index of i is subtracted when storing in w!
                if((int)fread( &(blob_data(w,out_depth,i-group_id*(b->d/conv_param->group),0)),sizeof(float),Ky*Kx, fp)!=Ky*Kx)
                    error("loading weights from file %s\n", conv_param->weights);

    //close file
    fclose(fp);

    //return weight blob
    return w;
}
void convolve_cpu(BLOB* in,BLOB* out,BLOB* w,int Kx,int Ky, conv_param_t* conv_param)
{
    for(int group_id=0;group_id<conv_param->group;group_id++)
            {
            int delta = (out->d/conv_param->group);//Depth of output divided by number of groups. 
            int output_starting_depth = group_id*delta;
            for(int out_depth=output_starting_depth;out_depth< output_starting_depth + delta;out_depth++)
                {
                int delta = (in->d/conv_param->group);//Depth of input divided by number of groups. 
                int in_depth_start = group_id*delta;
                for(int in_depth=in_depth_start;in_depth<in_depth_start+delta;in_depth++)
                    {
                    for(int out_y=0;out_y<out->h;out_y++)
                        for(int out_x=0;out_x<out->w;out_x++)
                            for(int ky=0;ky<Ky;ky++)
                                for(int kx=0;kx<Kx;kx++)
                                {
                                    int in_y = out_y*conv_param->Sy+ky;
                                    int in_x = out_x*conv_param->Sx+kx;

                                    int weigth_y = in_depth-(group_id*(in->d/conv_param->group));
                                    int weigth_x = ky*Kx + kx;

                                    float input = blob_data(in, in_depth, in_y,in_x);
                                    float weight = blob_data(w, out_depth, weigth_y, weigth_x);
                                     
                                    blob_data(out,out_depth,out_y,out_x)+= input*weight; 
                                }
                    }
                }
            }          


}


float* load_1d(const char* fname, size_t num){

    //open file for reading
    FILE* fp = fopen(fname, "rb");
    if(fp==NULL)
        error("could not open file %s for reading\n",fname);

    //read in array
    float* arr= (float*) malloc(sizeof(float)*num);
    if(fread(arr,sizeof(float), num, fp)!=num)
        error("loading data from file %s\n", fname);

    //close file
    fclose(fp);

    return arr;
}
blob_dims* get_gpu_blob_dims(BLOB* out,BLOB* weight,BLOB*in)
{
    int numblob_dims    = 3,
        blob_dim_size    = sizeof(blob_dims),
        numBytes     = numblob_dims * blob_dim_size;
        // allocate memory
        blob_dims *cpu_blob_dim_arr,*gpu_blob_dim_arr;
        cpu_blob_dim_arr = (blob_dims*)malloc(numBytes);
    
    hipMalloc((void**)&gpu_blob_dim_arr, numBytes);
    // 0 = out
    // 1 = weight
    // 2 = in
    cpu_blob_dim_arr[0].d = out->d;
    cpu_blob_dim_arr[0].w = out->w;
    cpu_blob_dim_arr[0].h = out->h;

    cpu_blob_dim_arr[1].d = weight->d;
    cpu_blob_dim_arr[1].w = weight->w;
    cpu_blob_dim_arr[1].h = weight->h;

    cpu_blob_dim_arr[2].d = in->d;
    cpu_blob_dim_arr[2].w = in->w;
    cpu_blob_dim_arr[2].h = in->h;
    hipMemcpy(gpu_blob_dim_arr,cpu_blob_dim_arr,numBytes,hipMemcpyHostToDevice);
    free(cpu_blob_dim_arr);
    return gpu_blob_dim_arr;
}

__device__ int calc_blob_id(int z,int y,int x,int height,int width)
{
    return z * height * width + y * width + x;

}

int cpu_calc_blob_id(int z,int y,int x,int height,int width)
{
    return z * height * width + y * width + x;

}


__global__ void gpu_device_convolve
    (float* data_in,float * data_weight, float* data_out // Data
    ,int Sx,int Sy // Sizes ...
    ,int in_w,int in_h,int in_d, // input blob dimensions
    int w_w,int w_h, // weigth height and depth
    int out_w,int out_h, // output width and height
    int Ky,int Kx 
    ,int in_depth,int out_depth,
    int group_id,int group){

    unsigned int out_x = blockIdx.x*blockDim.x + threadIdx.x;  
    unsigned int out_y = blockIdx.y*blockDim.y + threadIdx.y;
    if(out_x < out_w && out_y < out_h)//dims[0].w && out_y<dims[0].h)
    {
        int out_id = calc_blob_id(out_depth,out_y,out_x,out_h,out_w);
        for(int ky=0;ky<Ky;ky++)
        {
            for(int kx=0;kx<Kx;kx++)
            {
                int in_y = out_y*Sy+ky;
                int in_x = out_x*Sx+kx;

                int weigth_y = in_depth-(group_id*(in_d/group));
                int weight_x = ky*Kx + kx;
                
                int weight_id = calc_blob_id(out_depth,weigth_y,weight_x,w_h,w_w);
                int in_id = calc_blob_id(in_depth,in_y,in_x,in_h,in_w);
   
                data_out[out_id] += data_weight[weight_id] * data_in[in_id]; 
            }
        }
    }
}


__global__ void gpu_device_convolve_naive
    (float* data_in,float * data_weight, float* data_out // Data
    ,int Sx,int Sy // Sizes ...
    ,int in_w,int in_h,int in_d // input blob dimensions
    ,int w_w,int w_h // weigth height and depth
    ,int out_w,int out_h,int out_d // output width and height
    ,int Ky,int Kx 
    ,int out_depth_start
    ,int group_id,int group
    ,int in_delta,int in_depth_start)

    {
    unsigned int out_x = blockIdx.z*blockDim.z+ threadIdx.z;  
    unsigned int out_y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int out_depth = blockIdx.x*blockDim.x + threadIdx.x+ out_depth_start;
    
    if(out_depth < out_d && out_x < out_w && out_y < out_h)
    {
        int out_id = calc_blob_id(out_depth,out_y,out_x,out_h,out_w);
            
        for(int in_depth=in_depth_start;in_depth<in_depth_start+in_delta;in_depth++)
            {
                {
                    for(int ky=0;ky<Ky;ky++)
                    {
                        for(int kx=0;kx<Kx;kx++)
                        {
                            int in_y = out_y*Sy+ky;
                            int in_x = out_x*Sx+kx;

                            int weigth_y = in_depth-(group_id*(in_d/group));
                            int weight_x = ky*Kx + kx;
                            
                            int weight_id = calc_blob_id(out_depth,weigth_y,weight_x,w_h,w_w);
                            int in_id = calc_blob_id(in_depth,in_y,in_x,in_h,in_w);
            
                            data_out[out_id] += data_weight[weight_id] * data_in[in_id]; 
                        }
                    }
                }
        }
    }
}





void convolve_gpu(BLOB* in,BLOB* out,BLOB* w,int Kx,int Ky, conv_param_t* conv_param)
{

    

  
  //  printf("Width : %i ",out->w);
  //  printf(", Height : %i ",out->h);

    int in_delta = in->d/conv_param->group;//Depth of input divided by number of groups. 
    int out_delta = out->d/conv_param->group;//Depth of output divided by number of groups. 
    
  
    float* in_data;
    float* out_data;    
    float* w_data;
    
    blob2gpu(in_data, in);
    int last_id = out->w*out->h*out->d-1;
    blob2gpu(out_data, out);
    blob2gpu(w_data, w);
    blob_dims *gpu_blob_dim_arr = get_gpu_blob_dims(in,w,out);
    
  

    int numBlocksX=16;
    int numBlocksY=8;
    int numBlocksZ=8;
  

    int threadsPerBlockX = get_next_pow2(out_delta/numBlocksX+1);
    int threadsPerBlockY=get_next_pow2(out->h/numBlocksY+1);
    int threadsPerBlockZ=get_next_pow2(out->w/numBlocksZ+1);

    //printf(", out_delta : %i \n",out_delta);



    if(out_delta == 1)
    {
        numBlocksX = 1;
        threadsPerBlockX = 1;
    }
    if(out_delta == 96 && out->w == 112)
    {
        return convolve_cpu(in,out,w,Kx,Ky, conv_param);

    }
    
    dim3 grid( numBlocksX,numBlocksY, numBlocksZ );             // numBlocksX x numBlocksY ( x 1)
    dim3 block(threadsPerBlockX, threadsPerBlockY, threadsPerBlockZ);  // threadsPerBlockX x threadsPerBlockY x 3
    
   // dim3 grid( 4, 4, 8 );             // numBlocksX x numBlocksY ( x 1)
   // dim3 block(4, 4, 8);  // threadsPerBlockX x threadsPerBlockY x 3

   // printf("GRID : (x : %i) (y : % i) (z : %i) , ",numBlocksX,numBlocksY,numBlocksZ);
   // printf("BLOCK : (x : %i) (y : % i) (z : %i) \n",threadsPerBlockX,threadsPerBlockY,threadsPerBlockZ);

    for(int group_id=0;group_id<conv_param->group;group_id++)
    {
//                out_test<<<grid,block>>>(out_data,out_depth,gpu_blob_dim_arr);
        int out_depth_start = group_id*out_delta;
        int in_depth_start = group_id * in_delta;

                //printf("OutDepth : %i \n",out_depth );
        gpu_device_convolve_naive<<<grid,block>>>(
                in_data,w_data,out_data
                ,conv_param->Sx,conv_param->Sy
                ,in->w,in->h,in->d
                ,w->w,w->h
                ,out->w,out->h,out->d
                ,Ky,Kx
                ,out_depth_start
                ,group_id,conv_param->group
                ,in_delta,in_depth_start);
    }
        
              

    hipFree(gpu_blob_dim_arr);
     
 //   gpu2blob(in,in_data);
    gpu2blob(out,out_data);
    // gpu2blob(w,w_data);
    // hipFree(in_data);
    hipFree(out_data);

    // hipFree(w_data);
    // cudaCheckError();
    // cudaCheckError(hipFree(out_data));
    // cudaCheckError(hipFree(w_data));
    


}





BLOB* initialize_outputBlob(conv_param_t* conv_param,int height,int width)
{
    BLOB* out;
    if(conv_param->bias==NULL){
        //zero init
        out = blob_calloc(conv_param->num_out, height, width);
    }else{
        //not required to calloc
        out = blob_alloc(conv_param->num_out, height, width);

        //load bias values from file
        float* bias =load_1d(conv_param->bias, conv_param->num_out);

        //set bias or init with zeroes
        for(int out_depth=0;out_depth<out->d;out_depth++)
            for(int out_y=0;out_y<out->h;out_y++)
                for(int out_x=0;out_x<out->w;out_x++)
                    blob_data(out,out_depth,out_y,out_x)=bias[out_depth];

        //cleanup bias
        free(bias);
    }
    return out;

}

void printArrays(float * arr1,float * arr2)
{
    int i;
    for (i=0;i < sizeof(arr1) / sizeof(float);i++) {
        printf("%lf %lf\n",arr1[i],arr2[i]);
    }

}

void CompareBlobs(BLOB * blob,BLOB * blob2)
{
    for(int z = 0; z<blob->d;z++)
    {
        for(int y = 0; y<blob->h;y++)
        {
            for(int x = 0; x<blob->w;x++)
            {
                float delta = blob_data(blob,z,y,x) - blob_data(blob2,z,y,x) > 0.01;
                if(delta > 0.01 || delta < -0.01)
                {
                    printf("%lf Not Equal to %lf at x : %i , y : %i , z : %i \n"  ,blob_data(blob,z,y,x),blob_data(blob2,z,y,x),x,y,z);   
                    
                    x = y = z=100000;
                    break;
                
                }
            }   
        }   
    }

}


//convolution, NOTE: destructive of BLOB* in. duplicate if further required!
BLOB* convolution(BLOB* input, conv_param_t* conv_param){

    //use local pointer
    BLOB* in = input;

    //padding of input if required
    if(conv_param->pad!=0)
        in = pad(in, conv_param->pad);

    //if fully connected, the kernel size is set to the image size
    int Ky=(conv_param->fc)?in->h:conv_param->Ky;
    int Kx=(conv_param->fc)?in->w:conv_param->Kx;

    //create blob to hold output
    int height=(int)floor(((float)in->h - (float)Ky)/(float)conv_param->Sy)+1;
    int width =(int)floor(((float)in->w - (float)Kx)/(float)conv_param->Sx)+1;
    
    BLOB* out = initialize_outputBlob(conv_param,height,width);
    BLOB* out2 = initialize_outputBlob(conv_param,height,width);
    
    //load weightsint input_id = 
    BLOB* w = load_weights(in, conv_param);
    //convolve_gpu(in,out,w,Kx,Ky,conv_param);

//    convolve_cpu(in,out2,w,Kx,Ky,conv_param);
    convolve_gpu(in,out,w,Kx,Ky,conv_param);
//    CompareBlobs(out,out2);

//    printArrays(out->data,out2->data);

    //free weights
    blob_free(w);

    //done with padded blob, free
    if(conv_param->pad!=0)
        blob_free(in);

    //perform batchnorm if needed
    if(conv_param->bn_mean!=NULL){


        //load batchnorm mean and variance
        float* mean = load_1d(conv_param->bn_mean, out->d);
        float* var  = load_1d(conv_param->bn_var, out->d);

        //batchnorm
        for(int out_depth=0;out_depth<out->d;out_depth++)
            for(int out_y=0;out_y<out->h;out_y++)
                for(int out_x=0;out_x<out->w;out_x++)
                    blob_data(out,out_depth,out_y,out_x)= (blob_data(out,out_depth,out_y,out_x) - mean[out_depth])/sqrtf(var[out_depth]+conv_param->bn_eps);

        //free mean and variance
        free(mean);
        free(var);
    }

    //perform scale if needed
    if(conv_param->scale!=NULL){
        //load scale parameters
        float* scale = load_1d(conv_param->scale, out->d);
        float* scale_bias = load_1d(conv_param->scale_bias, out->d);

        //scale
        for(int out_depth=0;out_depth<out->d;out_depth++)
            for(int out_y=0;out_y<out->h;out_y++)
                for(int out_x=0;out_x<out->w;out_x++)
                    blob_data(out,out_depth,out_y,out_x) = blob_data(out,out_depth,out_y,out_x)*scale[out_depth] + scale_bias[out_depth];

        //free parameters
        free(scale);
        free(scale_bias);
    }

    //perform relu
    if(conv_param->relu==true)
        for(int i=0;i<blob_size(out); i++)
            out->data[i] =  fmax(0.0f, out->data[i]);

    //return output
    return out;
}
