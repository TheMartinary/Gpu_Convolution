#include "hip/hip_runtime.h"
#include "preprocessing.h"
#include "logging.h"

void cpu_preprocess(BLOB* img){
    //Subtract mean RGB values, scale with 0.017, and swap RGB->BGR
    for(int y=0;y<img->h;y++)
        for(int x=0;x<img->w;x++){
            float R              =  (blob_data(img,0,y,x)-123.680f)*0.017f; //R
            blob_data(img, 1,y,x) = (blob_data(img,1,y,x)-116.779f)*0.017f; //G
            blob_data(img, 0,y,x) = (blob_data(img,2,y,x)-103.939f)*0.017f; //B
            blob_data(img, 2,y,x) = R;
        }
}


__global__ void gpu_device_preprocess(float* data_in, float* data_out){

    unsigned int global_x = blockIdx.x*blockDim.x + threadIdx.x;  
    unsigned int global_y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int global_z = blockIdx.z*blockDim.z + threadIdx.z; //NOTE: gridDim.z==1 and thus blockIdx.z==0 in this example!

    unsigned int img_width  = gridDim.x*blockDim.x;
    unsigned int img_height = gridDim.y*blockDim.y;

       float value = data_in[ global_z*img_height*img_width + global_y*img_width + global_x];
    //each channel (Z) needs to correct with a different mean value
    float mean[3]={
        123.680f,
        116.779f,
        103.939f
    };

    //correct by subtracting the correct mean for this channel and scaling by a factor 0.017
    value= (value-mean[global_z]) * 0.017f;
    // swap rgb
    data_out[(2-global_z)*img_width*img_height + global_y*img_width + global_x]=value;
}

//GPU host code (called from the CPU, copies data back and forth and launched the GPU thread)
void gpu_preprocess(BLOB* img){

   
    //let's first divide the X and Y dimensions of the image into a number of blocks here
    int numBlocksX=16;
    int numBlocksY=16;

    int threadsPerBlockX=img->w/numBlocksX;  //NOTE: this should have remainder==0 for this code!!
    int threadsPerBlockY=img->h/numBlocksY;  //NOTE: this should have remainder==0 for this code!!

    dim3 grid( numBlocksX, numBlocksY, 1 );             // numBlocksX x numBlocksY ( x 1)
    dim3 block(threadsPerBlockX, threadsPerBlockY, 3);  // threadsPerBlockX x threadsPerBlockY x 3

    //Now that we have decided on the grid and block dimensions, it's time to copy our
    //image data over from the CPU to the GPUs global memory

    //First create a pointer to data on the GPU
    float* device_data;
#ifndef SHORTHANDS
    //This variable holds return values of cuda functions, which can be very useful for error checking
    hipError_t err;

    //malloc space on the on the GPU
    err=hipMalloc(&device_data, blob_bytes(img));

    //check for errors (NOTE: this is not a standard cuda function. Check logging.h)
    cudaCheckError(err)

    //copy the image data over to the GPU
    cudaCheckError(hipMemcpy(device_data, img->data, blob_bytes(img), hipMemcpyHostToDevice));
#else
    //For your convenience a helper function is defined in blob.h which can take care of the allocation and memcpy of blobs
    blob2gpu(device_data, img);
#endif

    //next we also allocate a buffer that will hold the output
    float* device_out;
    cudaCheckError(hipMalloc(&device_out, blob_bytes(img)));

    //Perform the preprocessing on the GPU
    info("Preprocessing on GPU...\n");
    gpu_device_preprocess<<< grid, block >>>(device_data, device_out);

    //We use "peekatlasterror" since a kernel launch does not return a hipError_t to check for errors
    cudaCheckError(hipPeekAtLastError());

#ifndef SHORTHANDS
    //copy the processed image data back from GPU global memory to CPU memory
    cudaCheckError(hipMemcpy(img->data, device_out, blob_bytes(img), hipMemcpyDeviceToHost));

    //free the allocated GPU memory that holds the output
    cudaCheckError(hipFree(device_out));
#else
    //again a simple shorthand to transfer a blob back from the gpu to the cpu and free the allocated memory
    gpu2blob(img, device_data);
#endif

    //finally we also need to release the space that holds the input on the GPU
    cudaCheckError(hipFree(device_data));

}
