#include "hip/hip_runtime.h"
#include "blob.h"
#include "convolution.h"

void convolve_cpu(BLOB* in,BLOB* out,BLOB* w,int Kx,int Ky, conv_param_t* conv_param)
{
    for(int group_id=0;group_id<conv_param->group;group_id++)
            {
            int delta = (out->d/conv_param->group);//Depth of output divided by number of groups. 
            int output_starting_depth = group_id*delta;
            for(int out_depth=output_starting_depth;out_depth< output_starting_depth + delta;out_depth++)
                {
                int delta = (in->d/conv_param->group);//Depth of input divided by number of groups. 
                int in_depth_start = group_id*delta;
                for(int in_depth=in_depth_start;in_depth<in_depth_start+delta;in_depth++)
                    {
                    for(int out_y=0;out_y<out->h;out_y++)
                        for(int out_x=0;out_x<out->w;out_x++)
                            for(int ky=0;ky<Ky;ky++)
                                for(int kx=0;kx<Kx;kx++)
                                {
                                    int in_y = out_y*conv_param->Sy+ky;
                                    int in_x = out_x*conv_param->Sx+kx;

                                    int weigth_y = in_depth-(group_id*(in->d/conv_param->group));
                                    int weigth_x = ky*Kx + kx;

                                    float input = blob_data(in, in_depth, in_y,in_x);
                                    float weight = blob_data(w, out_depth, weigth_y, weigth_x);
                                     
                                    blob_data(out,out_depth,out_y,out_x)+= input*weight; 
                                }
                    }
                }
            }          


}


__device__ int calc_blob_id(int z,int y,int x,int height,int width)
{
    return z * height * width + y * width + x;

}

__global__ void gpu_device_convolve_super_naive
    (float* data_in,float * data_weight, float* data_out // Data
    ,int Sx,int Sy // Sizes ...
    ,int in_w,int in_h,int in_d, // input blob dimensions
    int w_w,int w_h, // weigth height and depth
    int out_w,int out_h, // output width and height
    int Ky,int Kx 
    ,int in_depth,int out_depth,
    int group_id,int group){

    unsigned int out_x = blockIdx.x*blockDim.x + threadIdx.x;  
    unsigned int out_y = blockIdx.y*blockDim.y + threadIdx.y;
    if(out_x < out_w && out_y < out_h)//dims[0].w && out_y<dims[0].h)
    {
        int out_id = calc_blob_id(out_depth,out_y,out_x,out_h,out_w);
        for(int ky=0;ky<Ky;ky++)
        {
            for(int kx=0;kx<Kx;kx++)
            {
                int in_y = out_y*Sy+ky;
                int in_x = out_x*Sx+kx;

                int weigth_y = in_depth-(group_id*(in_d/group));
                int weight_x = ky*Kx + kx;
                
                int weight_id = calc_blob_id(out_depth,weigth_y,weight_x,w_h,w_w);
                int in_id = calc_blob_id(in_depth,in_y,in_x,in_h,in_w);
   
                data_out[out_id] += data_weight[weight_id] * data_in[in_id]; 
            }
        }
    }
}


__global__ void gpu_device_convolve_naive
    (float* data_in,float * data_weight, float* data_out // Data
    ,int Sx,int Sy // Sizes ...
    ,int in_w,int in_h,int in_d // input blob dimensions
    ,int w_w,int w_h // weigth height and depth
    ,int out_w,int out_h,int out_d // output width and height
    ,int Ky,int Kx 
    ,int out_depth_start
    ,int group_id,int group
    ,int in_delta,int in_depth_start)

    {
    unsigned int out_x = blockIdx.z*blockDim.z+ threadIdx.z;  
    unsigned int out_y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int out_depth = blockIdx.x*blockDim.x + threadIdx.x+ out_depth_start;
    
    if(out_depth < out_d && out_x < out_w && out_y < out_h)
    {
        int out_id = calc_blob_id(out_depth,out_y,out_x,out_h,out_w);
            
        for(int in_depth=in_depth_start;in_depth<in_depth_start+in_delta;in_depth++)
        {            
            for(int ky=0;ky<Ky;ky++)
            {
                for(int kx=0;kx<Kx;kx++)
                {
                    int in_y = out_y*Sy+ky;
                    int in_x = out_x*Sx+kx;

                    int weigth_y = in_depth-(group_id*(in_d/group));
                    int weight_x = ky*Kx + kx;
                    
                    int weight_id = calc_blob_id(out_depth,weigth_y,weight_x,w_h,w_w);
                    int in_id = calc_blob_id(in_depth,in_y,in_x,in_h,in_w);
    
                    data_out[out_id] += data_weight[weight_id] * data_in[in_id]; 
                }
            }        
        }
    }
}

int get_next_pow2(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;

}


void convolve_gpu(BLOB* in,BLOB* out,BLOB* w,int Kx,int Ky, conv_param_t* conv_param)
{

  int in_delta = in->d/conv_param->group;//Depth of input divided by number of groups. 
  int out_delta = out->d/conv_param->group;//Depth of output divided by number of groups. 

  float* in_data;
  float* out_data;    
  float* w_data;
  
  blob2gpu(in_data, in);
  blob2gpu(out_data, out);
  blob2gpu(w_data, w); 

  int numBlocksX=16;
  int numBlocksY=8;
  int numBlocksZ=8;

  int threadsPerBlockX = get_next_pow2(out_delta/numBlocksX+1);
  int threadsPerBlockY=get_next_pow2(out->h/numBlocksY+1);
  int threadsPerBlockZ=get_next_pow2(out->w/numBlocksZ+1);

  
  if(out_delta == 1)
  {
      numBlocksX = 1;
      threadsPerBlockX = 1;
  }
  if(out_delta == 96 && out->w == 112)
  {  // Cant get this specifc convolution to work
      return convolve_cpu(in,out,w,Kx,Ky, conv_param);

  }
  
  dim3 grid( numBlocksX,numBlocksY, numBlocksZ );          
  dim3 block(threadsPerBlockX, threadsPerBlockY, threadsPerBlockZ); 
  #ifdef DEBUG
  printf("out_width %i, out_height %i , out_delta : %i \n",out->w,out->h,out_delta);
  printf("in_width %i, in_height %i , in_delta : %i \n",in->w,in->h,in_delta);
  

  printf("GRID : (x : %i) (y : % i) (z : %i) , ",numBlocksX,numBlocksY,numBlocksZ);
  printf("BLOCK : (x : %i) (y : % i) (z : %i) \n",threadsPerBlockX,threadsPerBlockY,threadsPerBlockZ);
  #endif
  for(int group_id=0;group_id<conv_param->group;group_id++)
  {
      int out_depth_start = group_id*out_delta;
      int in_depth_start = group_id * in_delta;

     gpu_device_convolve_naive<<<grid,block>>>(
              in_data,w_data,out_data
              ,conv_param->Sx,conv_param->Sy
              ,in->w,in->h,in->d
              ,w->w,w->h
              ,out->w,out->h,out->d
              ,Ky,Kx
              ,out_depth_start
              ,group_id,conv_param->group
              ,in_delta,in_depth_start);
  }
   
  gpu2blob(out,out_data);

  hipFree(in_data);
  hipFree(out_data);
  hipFree(w_data);

}   